#include "hip/hip_runtime.h"
/*
Jueguito:

1) Sea h[i] el numero de granitos en el sitio i, 0<i<N-1.

2) Si h[i]>1 el sitio i esta "activo".

3) Al tiempo t, un sitio "activo" se "descarga" completamente tirando cada uno de sus granitos aleatoriamente y con igual probabilidad a la izquierda o a la derecha (el numero total de granitos entonces se conserva).

4) Los sitios se descargan sincronicamente. Entonces, a tiempo (t+1), el sitio activo i tendra h[i]=0 solo si sus vecinos no le tiraron granitos a tiempo t.

5) Se define la actividad A como el numero de sitios activos, es decir el numero de sitios que quieren descargarse.
Notar que si la densidad de granitos, [Suma_i h[i]/N] es muy baja, la actividad caera rapidamente a cero. Si la densidad es alta por otro lado, la actividad nunca cesara, ya que siempre habra sitios activos. En el medio hay una densidad "critica", para la cual la actividad decaera como una ley de potencia (pero se necesitaran sistemas grandes, y tiempos largos para verla bien definida).

*/

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <fstream>
#include <cstring>
#include <array>
#include <vector>
#include <cstdlib>
#include <random>
#include <cassert>

// number of sites
#define N (1024*1024) //TODO: se rompe todo si compilás con -DN=123, cambiar de N a NSLOTS o algo así
#define SIZE (N * 4)

#define BLOCK_SIZE 256

#define DENSITY 0.8924

// number of temporal steps
#define NSTEPS 10000

using namespace std;
typedef int * Manna_Array;

#define hiprandState hiprandStatePhilox4_32_10_t //fastest prng, tried XORWOW and MRG32k3a

__global__ void seedinit(hiprandState *seed, int first_num){
	hiprand_init(first_num,0,0,seed);
}

__global__ void randinit(hiprandState *seed, hiprandState *rand_state){
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(hiprand(seed),0,0,&rand_state[gtid]);
}

__device__ static inline bool randbool(hiprandState *rand_state){
	//~ return 1;
	return 1&hiprand(rand_state);
}

// CONDICION INICIAL ---------------------------------------------------------------
/*
Para generar una condicion inicial suficientemente uniforme con una densidad
lo mas aproximada (exacta cuando N->infinito) al numero real DENSITY, podemos hacer asi:
*/
__global__ void inicializacion(Manna_Array __restrict__ h)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	h[i] = (int)((i+1)*DENSITY)-(int)(i*DENSITY);
}

#ifdef DEBUG
void imprimir_array(Manna_Array __restrict__ h)
{
	int nrogranitos=0;
	int nrogranitos_activos=0;

	// esto dibuja los granitos en cada sitio y los cuenta
	for(int i = 0; i < N; ++i) {
		cout << h[i] << " ";
		nrogranitos += h[i];
		nrogranitos_activos += (h[i]>1);
	}
	cout << "\n";
	cout << "Hay " << nrogranitos << " granitos en total\n";
	cout << "De ellos " << nrogranitos_activos << " son activos\n";
	cout << "La densidad obtenida es " << nrogranitos*1.0/N;
	cout << ", mientras que la deseada era " << DENSITY << "\n\n";
}
#endif

__global__ void desestabilizacion_inicial(Manna_Array __restrict__ h, Manna_Array __restrict__ dh, unsigned int * __restrict__ slots_activos, hiprandState * __restrict__ rand_state)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (h[gtid]) {
		int k = (gtid+2*randbool(rand_state)-1+N)%N;
		//~ int k = (gtid+2*((gtid%3)%2)-1+N)%N; //trick to fix behavior
		atomicAdd(&dh[k], 1);
		h[gtid] = 0;
	}
}

__global__ void descargar(Manna_Array __restrict__ h, Manna_Array __restrict__ dh, unsigned int * __restrict__ slots_activos, hiprandState * __restrict__ rand_state)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	//~ unsigned int tid = threadIdx.x; // id hilo dentro del bloque
	//~ unsigned int lane = tid & CUDA_WARP_MASK; // id hilo dentro del warp, aka lane
	
	hiprandState *thread_state = &rand_state[gtid]; //doesn't get better if I use a local copy and then copy back
	
	// si es activo lo descargo aleatoriamente
	if (h[gtid] > 1) {
		for (int j = 0; j < h[gtid]; ++j) {
			int k = (gtid+2*randbool(thread_state)-1+N)%N;
			atomicAdd(&dh[k], 1);
		}
		h[gtid] = 0;
	}
	
	if(gtid==0) *slots_activos=0;
}

__global__ void actualizar(Manna_Array __restrict__ h, Manna_Array __restrict__ dh, unsigned int * __restrict__ result)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	h[gtid]+=dh[gtid];
	dh[gtid]=0; 	//zeroes dh array
	if(h[gtid]>1)
		atomicAdd(result, 1);
}

__device__ Manna_Array h,dh;
__device__ unsigned int slots_activos;

//===================================================================
int main(){
	ios::sync_with_stdio(0); cin.tie(0);
	assert(N%BLOCK_SIZE==0);
	
	//random initialization
	hiprandState *rand_state;
	hiprandState *seed;

	checkCudaErrors(hipMalloc(&rand_state, N*sizeof(hiprandState)));
	checkCudaErrors(hipMalloc(&seed, sizeof(hiprandState)));
	seedinit<<<1,1>>>(seed, time(NULL)); //initialize seed with some randomness
	getLastCudaError("seedinit failed");
	randinit<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(seed,rand_state); //initialize one state per thread based on previous random seed
	getLastCudaError("randinit failed");

	//slots
	checkCudaErrors(hipMalloc(&h, N*sizeof(int)));
	checkCudaErrors(hipMalloc(&dh, N*sizeof(int)));
	checkCudaErrors(hipMemset(dh, 0, N*sizeof(int)));

	//gets actual address in device (&slots_activos is garbage)
	unsigned int *slots_activos_addr;
	hipGetSymbolAddress((void **)&slots_activos_addr, slots_activos);

	//initialize slots
	cout << "estado inicial estable de la pila de arena...";
	inicializacion<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(h);
	getLastCudaError("inicializacion failed");
	cout << "LISTO\n";
	
	#ifdef DEBUG
	imprimir_array(h);
	#endif

	//create some chaos among slots
	cout << "estado inicial desestabilizado de la pila de arena...";
	desestabilizacion_inicial<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh,slots_activos_addr,rand_state);
	getLastCudaError("desestabilizacion failed");
	actualizar<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh,slots_activos_addr);
	getLastCudaError("actualizar failed");
	cout << "LISTO\n";
	
	#ifdef DEBUG
	imprimir_array(h);
	#endif

	cout << "evolucion de la pila de arena..."; cout.flush();

	ofstream activity_out("activity.dat");
	unsigned int activity;
	int t = 0;
	do {
		descargar<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh,slots_activos_addr,rand_state);
		getLastCudaError("descargar failed");
		actualizar<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh,slots_activos_addr);
		getLastCudaError("actualizar failed");
		checkCudaErrors(hipMemcpyFromSymbol(&activity, HIP_SYMBOL(slots_activos), sizeof(unsigned int)));
		
		activity_out << activity << "\n";
		#ifdef DEBUG
		imprimir_array(h);
		#endif
		++t;
	} while(activity > 0 && t < NSTEPS); // si la actividad decae a cero, esto no evoluciona mas...

	cout << "LISTO: " << ((activity>0)?("se acabo el tiempo\n\n"):("la actividad decayo a cero\n\n")); cout.flush();

	//free everything
	hipFree(h);
	hipFree(dh);
	hipFree(rand_state);
	hipFree(seed);

	return 0;
}

/*
 * TODO:
 * 		Try more work per thread. Change algorithm to get rid of many atomicAdd
 * 		make N and BLOCK_SIZE defineable during compile time
 * 		try normal distribution with: int hiprand_discrete(hiprandState_t *state, hiprandDiscreteDistribution_t discrete_distribution)
 */
