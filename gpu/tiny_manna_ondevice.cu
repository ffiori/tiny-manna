#include "hip/hip_runtime.h"
/*
Jueguito:

1) Sea h[i] el numero de granitos en el sitio i, 0<i<N-1.

2) Si h[i]>1 el sitio i esta "activo".

3) Al tiempo t, un sitio "activo" se "descarga" completamente tirando cada uno de sus granitos aleatoriamente y con igual probabilidad a la izquierda o a la derecha (el numero total de granitos entonces se conserva).

4) Los sitios se descargan sincronicamente. Entonces, a tiempo (t+1), el sitio activo i tendra h[i]=0 solo si sus vecinos no le tiraron granitos a tiempo t.

5) Se define la actividad A como el numero de sitios activos, es decir el numero de sitios que quieren descargarse.
Notar que si la densidad de granitos, [Suma_i h[i]/N] es muy baja, la actividad caera rapidamente a cero. Si la densidad es alta por otro lado, la actividad nunca cesara, ya que siempre habra sitios activos. En el medio hay una densidad "critica", para la cual la actividad decaera como una ley de potencia (pero se necesitaran sistemas grandes, y tiempos largos para verla bien definida).

*/

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <fstream>
#include <cstring>
#include <array>
#include <vector>
#include <cstdlib>
#include <random>
#include <cassert>

// number of sites
#define N (1024*1024) //TODO: se rompe todo si compilás con -DN=123, cambiar de N a NSLOTS o algo así
#define SIZE (N * 4)

#define BLOCK_SIZE 256

#define DENSITY 0.8924

// number of temporal steps
#define NSTEPS 10000

using namespace std;
typedef int * Manna_Array;

//fastest prng is XORWOW, default.
//~ #define hiprandState hiprandStatePhilox4_32_10_t 	//not so slow
//~ #define hiprandState hiprandStateMRG32k3a_t 		//slowest by far

__device__ hiprandState seed[1];
__device__ hiprandState rand_state[N];

__global__ void seedinit(int first_num){ //190ms, not top priority
	hiprand_init(first_num,0,0,seed);
	for(int i=0; i<N; i++) //must do it sequentially because of race conditions in hiprand(seed)
		hiprand_init(hiprand(seed),0,0,&rand_state[i]);
}

__device__ static inline bool randbool(hiprandState *rand_state){
	//~ return 1; //trick to fix behaviour
	return 1&hiprand(rand_state);
}

// CONDICION INICIAL ---------------------------------------------------------------
/*
Para generar una condicion inicial suficientemente uniforme con una densidad
lo mas aproximada (exacta cuando N->infinito) al numero real DENSITY, podemos hacer asi:
*/
__global__ void inicializacion(Manna_Array __restrict__ h)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	h[i] = (int)((i+1)*DENSITY)-(int)(i*DENSITY);
}

__global__ void desestabilizacion_inicial(Manna_Array __restrict__ h, Manna_Array __restrict__ dh)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (h[gtid]) {
		int k = (gtid+2*randbool(&rand_state[gtid])-1+N)%N;
		//~ int k = (gtid+2*((gtid%3)%2)-1+N)%N; //trick to fix behavior
		atomicAdd(&dh[k], 1);
		h[gtid] = 0;
	}
}

__device__ unsigned int *activity;
__device__ unsigned int slots_activos;
unsigned int activity_host[NSTEPS+1];

__global__ void descargar(Manna_Array __restrict__ h, Manna_Array __restrict__ dh, int t, unsigned int *activity)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	//~ unsigned int tid = threadIdx.x; // id hilo dentro del bloque
	//~ unsigned int lane = tid & CUDA_WARP_MASK; // id hilo dentro del warp, aka lane
	//~ uint warp = tid / CUDA_WARP_SIZE;  // warp dentro del bloque
	//~ uint gwarp = gtid / CUDA_WARP_SIZE;  // Identificador global de warp
	//~ uint bid = blockIdx.x;  // Identificador de bloque
	
	hiprandState *thread_state = &rand_state[gtid]; //doesn't get better if I use a local copy and then copy back
	
	if (h[gtid] > 1) {
		for (int j = 0; j < h[gtid]; ++j) {
			int k = (gtid+2*randbool(thread_state)-1+N)%N;
			atomicAdd(&dh[k], 1);
		}
	} else atomicAdd(&dh[gtid], h[gtid]);
	h[gtid] = 0;

	if(gtid==0) {
		activity[t] = slots_activos;
		slots_activos=0;
	}
}

__global__ void actualizar(Manna_Array __restrict__ h, Manna_Array __restrict__ dh, int t)
{
	unsigned int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	if(h[gtid]>1)
		atomicAdd(&slots_activos, 1);
}

__device__ Manna_Array h,dh;

//===================================================================
int main(){
	ios::sync_with_stdio(0); cin.tie(0);
	assert(N%BLOCK_SIZE==0);
	
	//random initialization
	seedinit<<<1,1>>>(time(NULL)); //initialize a state per thread with some random seed
	getLastCudaError("seedinit failed");

	//slots
	checkCudaErrors(hipMalloc(&h, N*sizeof(int)));
	checkCudaErrors(hipMalloc(&dh, N*sizeof(int)));
	checkCudaErrors(hipMalloc(&activity, (NSTEPS+1)*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(dh, 0, N*sizeof(int)));

	//gets actual address in device (&slots_activos is garbage)
	unsigned int *slots_activos_addr;
	hipGetSymbolAddress((void **)&slots_activos_addr, slots_activos);

	//initialize slots
	cout << "estado inicial estable de la pila de arena...";
	inicializacion<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(h);
	getLastCudaError("inicializacion failed");
	cout << "LISTO\n";
	
	//create some chaos among slots
	cout << "estado inicial desestabilizado de la pila de arena...";
	desestabilizacion_inicial<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh);
	getLastCudaError("desestabilizacion failed");
	swap(h,dh);
	cout << "LISTO\n";
	
	cout << "evolucion de la pila de arena..."; cout.flush();

	ofstream activity_out("activity.dat");
	int t = 0;
	do {
		descargar<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh, t, activity);
		getLastCudaError("descargar failed");
		swap(h,dh);
		actualizar<<< N/BLOCK_SIZE, BLOCK_SIZE >>>(h,dh, t);
		getLastCudaError("actualizar failed");
		++t;
	} while(t < NSTEPS);
	
	checkCudaErrors(hipMemcpy(activity_host, activity, sizeof(activity_host), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&activity_host[NSTEPS], slots_activos_addr, sizeof(unsigned int), hipMemcpyDeviceToHost));

	bool timeout = true;
	for (int i = 1; i <= NSTEPS; i++) {
		activity_out << activity_host[i] << "\n";
		if (!activity_host[i]) { timeout = false; cout << "En i " << i << endl; break;}
	}

	cout << "LISTO: " << ((timeout)?("se acabo el tiempo\n\n"):("la actividad decayo a cero\n\n")); cout.flush();
	
	//free everything
	hipFree(h);
	hipFree(dh);
	hipFree(activity);

	return 0;
}

/*
 * TODO:
 * 		make N and BLOCK_SIZE defineable during compile time
 */
